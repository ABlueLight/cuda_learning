#include "hip/hip_runtime.h"
#include <stdio.h>  
#include <stdlib.h>  
#include <hip/hip_runtime.h> 

#define w 8000

//设备端代码
__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements)  
{  
  int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements)
    {
        C[i] = A[i] * B[i];
    }
}  


__global__ void DotMulVet(const float* A,const float* B,float* C,int N)
{    
    int index = blockIdx.x*blockDim.x+threadIdx.x;    
    const int offset = gridDim.x * blockDim.x;
 
 
    while (index < N)
    {
        C[index] = A[index] * B[index];
        index += offset;
    }
}





//主机端代码
extern "C" int func() // 注意这里定义形式  
{
  // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = 512;
    size_t size = numElements * sizeof(float);
    printf("[Vector addition of %d elements]\n", numElements);

    // Allocate the host input vector A
    float *h_A = (float *)malloc(size);

    // Allocate the host input vector B
    float *h_B = (float *)malloc(size);

    // Allocate the host output vector C
    float *h_C = (float *)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    printf("Index    h_A       h_B\n");
    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    	//printf("Index %d: %f  %f\n",i,h_A[i],h_B[i]);
    }
    printf("\n");

    // Allocate the device input vector A
    float *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);//分配一维的线性存储空间

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector B
    float *d_B = NULL;
    err = hipMalloc((void **)&d_B, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector C
    float *d_C = NULL;
    err = hipMalloc((void **)&d_C, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);//将一维线性存储器的数据从主机端传输到设备端

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    DotMulVet<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Verify that the result vector is correct
    for (int i = 0; i < 5; ++i)
    {
        if (fabs(h_A[i] * h_B[i] - h_C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

    printf("Test PASSED\n\n");

    printf("vectorAdd_Result:\n");
    for(int i=0;i<5;i++)
      printf("Index %d: %f\n",i,h_C[i]);
    printf("\n");
    // Free device global memory
    err = hipFree(d_A);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_B);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_C);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    printf("Done\n");
    return 0;
}

struct Matrix
{
    int width;
    int height;
    float *elements;
};


__device__ float getElement(Matrix *A, int row, int col)
{
        return A->elements[row * A->width + col];
}

__device__ void setElement(Matrix *A, int row, int col, float value)
{
        A->elements[row * A->width + col] = value;
}

__global__ void matMulKernel(Matrix *A, Matrix *B, Matrix *C)
{
        float Cvalue = 0.0;
        int row = threadIdx.y + blockIdx.y * blockDim.y;
        int col = threadIdx.x + blockIdx.x * blockDim.x;
        
        for (int i = 0; i < A->width; ++i)
        {
                Cvalue += getElement(A, row, i) * getElement(B, i, col);
        }
        setElement(C, row, col, Cvalue);
}

extern "C" int testmatMul() {
     int width = w;
    int height = w;

    Matrix *A, *B, *C;

    hipMallocManaged((void**)&A, sizeof(Matrix));
    hipMallocManaged((void**)&B, sizeof(Matrix));
    hipMallocManaged((void**)&C, sizeof(Matrix));

    int nBytes = width * height * sizeof(float);

    hipMallocManaged((void**)&A->elements, nBytes);
    hipMallocManaged((void**)&B->elements, nBytes);
    hipMallocManaged((void**)&C->elements, nBytes);

    A->height = height;
    A->width = width;
    B->height = height;
    B->width = width;
    C->height = height;
    C->width = width;

    for (int i = 0; i < width * height; ++i)
    {
        A->elements[i] = 1.0;
        B->elements[i] = 2.0;
    }

    dim3 blockSize(32, 32);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
        (height + blockSize.y - 1) / blockSize.y);

/*
    struct timeval t1,t2;
    gettimeofday(&t1,NULL);
    double timeuse;
*/
    matMulKernel << < gridSize, blockSize >> >(A, B, C);

    hipDeviceSynchronize();

/*
    gettimeofday(&t2,NULL);
    timeuse = t2.tv_sec - t1.tv_sec + (t2.tv_usec - t1.tv_usec)/1000000.0;
    printf("Use Time:%fs\n", timeuse);
*/
}